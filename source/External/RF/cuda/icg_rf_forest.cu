#ifndef ICG_RF_FOREST_CU_
#define ICG_RF_FOREST_CU_

#include "icg_rf_forest_kernels.cu"
#include "icg_rf_rng.cu"

#include "cudatemplates/hostmemoryheap.hpp" 

//#define DEBUG 

//-----------------------------------------------------------------------------
template <typename T>
void printMemUsage(const Cuda::DeviceMemoryLinear<T, 1> &mat, const char *name = ""){
    printf("DeviceMemoryPitched %s (%d x 1) needs %d KB of device memory\n", name, mat.size[0], mat.size[0] * sizeof(T)/1024);
}
template <typename T>
void printMemUsage(const Cuda::DeviceMemoryPitched<T, 2> &mat, const char *name = ""){
    printf("DeviceMemoryPitched %s (%d x %d) needs %d KB of device memory\n", name, mat.size[0], mat.size[1], mat.stride[0] * mat.size[1] * sizeof(T)/1024);
}
template <typename T>
void printMemUsage(const Cuda::Array<T, 1> &mat, const char *name = ""){
    printf("Array %s (%d x %d) needs %d KB of device memory\n", name, mat.size[0], mat.size[1], mat.size[0] * mat.size[1] * sizeof(T)/1024);
}
template <typename T>
void printMemUsage(const Cuda::Array<T, 2> &mat, const char *name = ""){
    printf("Array %s (%d x %d) needs %d KB of device memory\n", name, mat.size[0], mat.size[1], mat.size[0] * mat.size[1] * sizeof(T)/1024);
}


//-----------------------------------------------------------------------------
inline bool printToScreen(const Cuda::HostMemoryHeap<float, 1> &data, const char *name = ""){
    printf("\nprinting Matrix: %s (%d)\n", name, (int)data.size[0]);
    for (unsigned int x = 0; x < data.size[0]; x++)
        printf("%3.3f ", data.getBuffer()[x]);
    printf("\n");
    return true;
}
bool printToScreen(const Cuda::HostMemoryHeap<float, 2> &data, const char *name = ""){
    printf("\nprinting Matrix: %s (%d x %d)\n", name, (int)data.size[0], (int)data.size[1]);
    for (unsigned int y = 0; y < data.size[1]; y++){
        for (unsigned int x = 0; x < data.size[0]; x++){
            printf("%3.3f ", data.getBuffer()[y * data.stride[0] + x]);
        }
        printf("\n");
    }
    return true;
} 
bool printToScreen(const Cuda::HostMemoryHeap<int, 2> &data, const char *name = ""){
    printf("\nprinting Matrix: %s (%d x %d)\n", name, (int)data.size[0], (int)data.size[1]);
    for (unsigned int y = 0; y < data.size[1]; y++){
        for (unsigned int x = 0; x < data.size[0]; x++){
            printf("%d ", data.getBuffer()[y * data.stride[0] + x]);
        }
        printf("\n");
    }
    return true;
} 
bool printToScreen(const Cuda::HostMemoryHeap<unsigned char, 2> &data, const char *name = ""){
    printf("\nprinting Matrix: %s (%d x %d)\n", name, (int)data.size[0], (int)data.size[1]);
    for (unsigned int y = 0; y < data.size[1]; y++){
        for (unsigned int x = 0; x < data.size[0]; x++){
            printf("%d ", data.getBuffer()[y * data.stride[0] + x]);
        }
        printf("\n");
    }
    return true;
} 
bool printToScreen(const Cuda::DeviceMemoryPitched<int, 2> &data, const char *name = ""){
    Cuda::HostMemoryHeap<int, 2> data_copy(data);
    return printToScreen(data_copy, name);
}
bool printToScreen(const Cuda::DeviceMemoryPitched<unsigned char, 2> &data, const char *name = ""){
    Cuda::HostMemoryHeap<unsigned char, 2> data_copy(data);
    return printToScreen(data_copy, name);
}
bool printToScreen(const Cuda::DeviceMemoryPitched<float, 2> &data, const char *name = ""){
    Cuda::HostMemoryHeap<float, 2> data_copy(data);
    return printToScreen(data_copy, name);
}
//-----------------------------------------------------------------------------
bool printToScreen(const float* devptr, const int width, const int height, const int stride, const char *name = ""){
    Cuda::HostMemoryHeap<float,2> data(Cuda::Size<2>(width, height));
    //printf("\ncopying %d bytes in %d rows, stride = %d\n", width * sizeof(float), height, stride * sizeof(float));
    hipMemcpy2D(data.getBuffer(), width * sizeof(float),
        devptr, stride * sizeof(float),
        width * sizeof(float), height,
        hipMemcpyDeviceToHost);
    ICG_RF_CHECK_CUDA_ERROR();
    return printToScreen(data, name);
}

//-----------------------------------------------------------------------------
template <typename T>
inline bool cudaInit(Cuda::DeviceMemoryPitched<T,2>* data, T value){
    int bs = 16; 
    icg_rf_init_kernel<<<dim3(divUp(data->size[0], bs), divUp(data->size[1], bs),1), dim3(bs, bs, 1)>>>(
        data->getBuffer(),data->size[0], data->size[1],data->stride[0], value);
    ICG_RF_CHECK_CUDA_ERROR();
    return true;
}
//-----------------------------------------------------------------------------
template <typename T>
inline bool cudaInit(Cuda::HostMemoryHeap<T,2>* data, T value){
    for (size_t xy = 0; xy < data->size[0] * data->size[1]; xy++)
        data->getBuffer()[xy] = value;
    return true;
}

//-----------------------------------------------------------------------------
template <typename T>
inline T cudaGet(const Cuda::HostMemoryHeap<T,2> data, int x, int y){
#ifdef DEBUG
    if (x < 0 || x >= data.size[0] || y < 0 || y >= data.size[1])
        printf("\n !!! trying to access %d|%d in %d|%d\n", x,y,data.size[0], data.size[1]);
#endif
    return data.getBuffer()[y * data.stride[0] + x];          
}

//-----------------------------------------------------------------------------
template <typename T>
inline void cudaSet(Cuda::HostMemoryHeap<T,2> *data, int x, int y, T value){
#ifdef DEBUG
    if (x < 0 || x >= data->size[0] || y < 0 || y >= data->size[1])
        printf("\n !!! trying to write to %d|%d in %d|%d\n", x,y,data->size[0], data->size[1]);
#endif
    data->getBuffer()[y * data->stride[0] + x] = value;
}


//-----------------------------------------------------------------------------
bool cudaTrainForest(const Cuda::Array<float,2>& input,
                     const Cuda::DeviceMemoryLinear<int,1>& labels,
                     const Cuda::DeviceMemoryLinear<float,1>& weights,
                     Cuda::DeviceMemoryPitched<float,2>** forest,
                     float* oobe,
                     const int num_samples, const int num_classes, 
                     const int num_trees, const int max_depth, const int num_tree_cols,
                     const int num_hypotheses, const int num_features,
                     const float bag_ratio)
{ 
    // new random seed...
#ifdef WIN32
    unsigned int seedNum = (unsigned int) time(NULL);
    //    srand(seedNum); //FIXXME, only for reproducability
#else
    unsigned int seedNum;
    struct timeval TV;
    unsigned int curTime;
    gettimeofday(&TV, NULL);
    curTime = (unsigned int) TV.tv_usec;
    seedNum = (unsigned int) time(NULL) + curTime + getpid() + getDevRandom();
    srand(seedNum);        
#endif

    // Copy Training Parameters
    TrainParams tp;
    tp.num_trees = num_trees;
    tp.tree_level = 0;
    tp.max_depth = max_depth;
    tp.num_samples = num_samples;
    tp.num_features = num_features;
    tp.num_classes = num_classes;
    tp.num_hyp = num_hypotheses;
    tp.num_hyp_2 = num_hypotheses * num_hypotheses;
    tp.num_tree_cols = num_tree_cols;

    // Init textures
    hipChannelFormatDesc channelDescInt = hipCreateChannelDesc<int>();
    hipChannelFormatDesc channelDescFloat = hipCreateChannelDesc<float>();

    input_tex.filterMode = hipFilterModePoint;
    input_tex.normalized = false;
    hipBindTextureToArray(input_tex, input.getArray(), channelDescFloat);

    hipBindTexture(0, labels_tex, labels.getBuffer(), labels.size[0] * sizeof(int));

    r_weights_tex.filterMode = hipFilterModePoint;
    r_weights_tex.normalized = false;

    r_features_tex.filterMode = hipFilterModePoint;
    r_features_tex.normalized = false;

    // perform bagging
    int num_bag_samples = (int)(input.size[1] * bag_ratio);
    Cuda::DeviceMemoryPitched<int,2>bags_dmp(Cuda::Size<2>(num_trees, num_bag_samples));
    cudaRandomNumbers(&bags_dmp, 0, labels.size[0]); 

    // structure to store the node index in every tree that each sample is voting for currently
    Cuda::DeviceMemoryPitched<int,2> affiliation_dmp(Cuda::Size<2>(num_trees, num_samples));
    cudaInit(&affiliation_dmp, -1);

    // store 0 in affiliation, when sample is in bag
    int bs = 16;
    icg_rf_bagging_kernel<<<dim3(divUp(num_trees, bs), divUp(num_bag_samples, bs),1), dim3(bs, bs, 1)>>>(
        bags_dmp.getBuffer(),bags_dmp.stride[0],
        affiliation_dmp.getBuffer(), affiliation_dmp.stride[0],
        num_trees, num_bag_samples); 

    // Init trees 
    Cuda::HostMemoryHeap<float*, 2> tree_adress_hmh(Cuda::Size<2>(num_trees,1));
    size_t single_tree_pitch;
    for (int tree = 0; tree < num_trees; tree++) {
        hipMallocPitch((void**)&tree_adress_hmh.getBuffer()[tree], 
            &single_tree_pitch, num_tree_cols*sizeof(float), 1);        
        ICG_RF_CHECK_CUDA_ERROR();
        // init mem with zeros
        hipMemset2D(tree_adress_hmh.getBuffer()[tree], single_tree_pitch, 0, num_tree_cols*sizeof(float), 1);
        ICG_RF_CHECK_CUDA_ERROR();
    }
    single_tree_pitch /= sizeof(float);
    // Copy tree adresses to device
    Cuda::DeviceMemoryPitched<float*, 2> tree_adress_dmp(tree_adress_hmh);

    // Init structure that stores the number nodes in each level of a tree
    Cuda::DeviceMemoryPitched<int,2> num_nodes_per_level_dmp(Cuda::Size<2>(num_trees,max_depth + 1)); 
    cudaInit(&num_nodes_per_level_dmp, 1);
    Cuda::HostMemoryHeap<int,2> nodes_per_level_hmh(num_nodes_per_level_dmp);

    // store the maximum number of new nodes over all trees
    int num_leaf_nodes_max = 1;

    for (int tree_level = 0; tree_level <= max_depth; tree_level++) {
        tp.tree_level = tree_level;

        // classify nodes to leafs and splits  
        Cuda::DeviceMemoryPitched<int,2>node_type_dmp(Cuda::Size<2>(num_trees, num_leaf_nodes_max));
        if (num_leaf_nodes_max) {
            cudaInit(&node_type_dmp, -1);
            bs = 8;
            icg_rf_classify_leaf_nodes_kernel<<<dim3(divUp(num_trees, bs), divUp(num_leaf_nodes_max, bs),1), dim3(bs,bs,1)>>>(
                num_nodes_per_level_dmp.getBuffer(), num_nodes_per_level_dmp.stride[0],
                affiliation_dmp.getBuffer(), affiliation_dmp.stride[0],
                node_type_dmp.getBuffer(), node_type_dmp.stride[0],tp);
            ICG_RF_CHECK_CUDA_ERROR();
        }

        // find out the number of leaf and split nodes per tree and in total
        Cuda::HostMemoryHeap<int,2>node_type_hmh(node_type_dmp);
        Cuda::HostMemoryHeap<int,2>num_splits_hmh(Cuda::Size<2>(num_trees,1));
        Cuda::HostMemoryHeap<int,2>num_leafs_hmh(Cuda::Size<2>(num_trees,1));
        cudaInit(&num_splits_hmh, 0);
        cudaInit(&num_leafs_hmh, 0);
        int num_splits = 0;
        int num_leafs = 0;
        for (int tree = 0; tree < num_trees; tree++) {
            for (int node = 0; node < num_leaf_nodes_max; node++){
                if (cudaGet(node_type_hmh, tree, node) == 1){
                    num_leafs_hmh.getBuffer()[tree]++;
                    num_leafs++;
                }
                if (cudaGet(node_type_hmh, tree, node) == 2){
                    num_splits_hmh.getBuffer()[tree]++;
                    num_splits++;
                }
            }
        }

        // create subsets for the split and leaf nodes
        Cuda::HostMemoryHeap<int,2>split_subset_hmh(Cuda::Size<2>(num_splits, 3));
        Cuda::HostMemoryHeap<int,2>leaf_subset_hmh(Cuda::Size<2>(num_leafs, 2));
        cudaInit(&split_subset_hmh, 0);
        cudaInit(&leaf_subset_hmh, 0);
        int split_index = 0;
        int leaf_index = 0;
        for (int tree = 0; tree < num_trees; tree++) {
            int num_leaf_nodes = 1;
            int leaf_row_offset = 0;
            int left_son = cudaGet(nodes_per_level_hmh, tree, tree_level); 
            if (tree_level > 0) {
                leaf_row_offset = cudaGet(nodes_per_level_hmh, tree, tree_level-1);
                num_leaf_nodes = cudaGet(nodes_per_level_hmh, tree, tree_level)- leaf_row_offset; 
            }
            for (int node = 0; node < num_leaf_nodes; node++){
                if (cudaGet(node_type_hmh, tree, node) == 1){
                    cudaSet(&leaf_subset_hmh, leaf_index, 0, tree);
                    cudaSet(&leaf_subset_hmh, leaf_index++, 1, node + leaf_row_offset);
                }
                if (cudaGet(node_type_hmh, tree, node) == 2){
                    cudaSet(&split_subset_hmh, split_index, 0, tree);
                    cudaSet(&split_subset_hmh, split_index, 1, node + leaf_row_offset);
                    cudaSet(&split_subset_hmh, split_index++, 2, left_son);
                    left_son += 2;
                }
            }
        }
        Cuda::DeviceMemoryPitched<int,2>split_subset_dmp(split_subset_hmh);
        Cuda::DeviceMemoryPitched<int,2>leaf_subset_dmp(leaf_subset_hmh);

        // train the split nodes
        if (num_splits) {
            // create random numbers
            Cuda::DeviceMemoryPitched<int,2> rand_features_dmp(Cuda::Size<2>(num_splits, num_features));
            Cuda::DeviceMemoryPitched<float,2> rand_weights_dmp(Cuda::Size<2>(num_splits, num_features * num_hypotheses)); 
            cudaRandomNumbers(&rand_features_dmp, 0, input.size[0] - 1);
            cudaRandomNumbers(&rand_weights_dmp);

            // copy them to arrays            
            Cuda::Array<float,2> rand_weights_arr(rand_weights_dmp);
            Cuda::Array<int,2> rand_features_arr(rand_features_dmp);
            hipBindTextureToArray(r_weights_tex, rand_weights_arr.getArray(), channelDescFloat);
            hipBindTextureToArray(r_features_tex, rand_features_arr.getArray(), channelDescInt);
            ICG_RF_CHECK_CUDA_ERROR(); 

            // assign the features to the trees
            bs = 8;
            icg_rf_split_assign_feature_kernel<<<dim3(divUp(num_splits, bs), divUp(num_features, bs), 1), dim3(bs,bs,1)>>>(
                tree_adress_dmp.getBuffer(), single_tree_pitch,
                split_subset_dmp.getBuffer(), num_splits, split_subset_dmp.stride[0], tp);
            ICG_RF_CHECK_CUDA_ERROR();  

            // create buffer structures for training
            Cuda::DeviceMemoryPitched<float,2> thresh_limits_dmp(Cuda::Size<2>(num_splits, num_hypotheses * 2));
            Cuda::DeviceMemoryPitched<float,2> pk_l_dmp(Cuda::Size<2>(num_splits, tp.num_hyp_2 * num_classes));
            Cuda::DeviceMemoryPitched<float,2> pk_r_dmp(Cuda::Size<2>(num_splits, tp.num_hyp_2 * num_classes));
            cudaInit(&pk_l_dmp, 0.0f);
            cudaInit(&pk_r_dmp, 0.0f);

            // compute the minimum and maximum response for each node
            bs = 8;
            icg_rf_split_compute_thresh_kernel<<<dim3(divUp(num_splits, bs), divUp(num_hypotheses, bs), 1), dim3(bs,bs,1)>>>(
                split_subset_dmp.getBuffer(), num_splits, split_subset_dmp.stride[0],
                affiliation_dmp.getBuffer(), affiliation_dmp.stride[0],
                thresh_limits_dmp.getBuffer(), thresh_limits_dmp.stride[0],tp);

            // evaluate all hypotheses
            bs = 8;   
            icg_rf_split_eval_thresh_kernel<<<dim3(divUp(num_splits, bs), divUp(num_hypotheses, bs), 1), dim3(bs, bs, 1)>>>(
                split_subset_dmp.getBuffer(), num_splits, split_subset_dmp.stride[0],
                weights.getBuffer(), affiliation_dmp.getBuffer(), affiliation_dmp.stride[0],
                thresh_limits_dmp.getBuffer(), thresh_limits_dmp.stride[0],
                pk_l_dmp.getBuffer(), pk_l_dmp.stride[0], pk_r_dmp.getBuffer(), pk_r_dmp.stride[0],tp);
            ICG_RF_CHECK_CUDA_ERROR();

            // select best hypothesis
            bs = 64;
            icg_rf_split_select_best_kernel<<<dim3(divUp(num_splits, bs), 1, 1), dim3(bs, 1, 1)>>>(
                tree_adress_dmp.getBuffer(), single_tree_pitch,
                split_subset_dmp.getBuffer(), num_splits, split_subset_dmp.stride[0],
                thresh_limits_dmp.getBuffer(), thresh_limits_dmp.stride[0],
                pk_l_dmp.getBuffer(), pk_l_dmp.stride[0], pk_r_dmp.getBuffer(), pk_r_dmp.stride[0], tp);
            ICG_RF_CHECK_CUDA_ERROR();

            hipUnbindTexture(r_weights_tex);
            hipUnbindTexture(r_features_tex);
        }
        // train leaf nodes
        if (num_leafs) {
            bs = 32;
            size_t shared_mem = bs * num_classes * sizeof(float);
            if (shared_mem > 16000)
                ICG_RF_THROW_ERROR("icg_rf_train_leaf_node_kernel: too much shared memory needed...");
            icg_rf_train_leaf_node_kernel<<<dim3(divUp(num_leafs, bs), 1, 1), dim3(bs, 1, 1), shared_mem>>>(
                tree_adress_dmp.getBuffer(), single_tree_pitch, weights.getBuffer(),
                leaf_subset_dmp.getBuffer(), num_leafs, leaf_subset_dmp.stride[0],
                affiliation_dmp.getBuffer(), affiliation_dmp.stride[0], tp);
            ICG_RF_CHECK_CUDA_ERROR(); 
        }  

        // update trees for the next iteration
        num_leaf_nodes_max = 0;
        if (tree_level < max_depth) {
            // reset the number of rows per tree
            for (int tree = 0; tree < num_trees; tree++) {
                int num_new_rows = num_splits_hmh.getBuffer()[tree] * 2;
                cudaSet(&nodes_per_level_hmh, tree, tree_level+1, cudaGet(nodes_per_level_hmh, tree, tree_level) + num_new_rows);
                if (num_new_rows > num_leaf_nodes_max)
                    num_leaf_nodes_max = num_new_rows; 
            }
            Cuda::copy(num_nodes_per_level_dmp, nodes_per_level_hmh);   

            // create new trees and copy the old ones        
            for (int tree = 0; tree < num_trees; tree++) {
                int num_new_rows = num_splits_hmh.getBuffer()[tree] * 2;  
                if (num_new_rows) {
                    float* new_tree;
                    size_t new_pitch;
                    hipMallocPitch((void**)&new_tree, &new_pitch, num_tree_cols*sizeof(float), 
                        cudaGet(nodes_per_level_hmh, tree, tree_level+1));        
                    // init mem with zeros
                    hipMemset2D(new_tree, new_pitch, 0, num_tree_cols*sizeof(float), 
                        cudaGet(nodes_per_level_hmh, tree, tree_level+1));
                    // copy old tree inside
                    hipMemcpy2D(new_tree, new_pitch, 
                        tree_adress_hmh.getBuffer()[tree], single_tree_pitch*sizeof(float),
                        num_tree_cols*sizeof(float), cudaGet(nodes_per_level_hmh, tree, tree_level), 
                        hipMemcpyDeviceToDevice);
                    ICG_RF_CHECK_CUDA_ERROR();
                    hipFree(tree_adress_hmh.getBuffer()[tree]);
                    tree_adress_hmh.getBuffer()[tree] = new_tree;
                }
            }
            Cuda::copy(tree_adress_dmp, tree_adress_hmh);
        }

        // update affiliation (node index that each sample gets stuck to)
        bs = 8;
        icg_rf_eval_affinity_kernel<<<dim3(divUp(num_samples, bs), divUp(num_trees, bs),1), dim3(bs, bs, 1)>>>( 
            tree_adress_dmp.getBuffer(), single_tree_pitch,
            affiliation_dmp.getBuffer(), affiliation_dmp.stride[0],tp);
        ICG_RF_CHECK_CUDA_ERROR();
    } 

    // find out the maximum number of rows in the forest
    int max_rows = 0;
    for (int tree = 0; tree < num_trees; tree++) {
        int num_rows = cudaGet(nodes_per_level_hmh, tree, max_depth);
        if (num_rows > max_rows)
            max_rows = num_rows;
    }
    // create forest structure
    *forest = new Cuda::DeviceMemoryPitched<float,2>(Cuda::Size<2>(num_tree_cols * num_trees, max_rows));
    ICG_RF_CHECK_CUDA_ERROR();

    // compute oobe
    /*Cuda::DeviceMemoryPitched<int, 2> oobe_prediction_dmp(Cuda::Size<2>(num_trees, num_samples));
    cudaInit(&oobe_prediction_dmp, -1);
    ICG_RF_CHECK_CUDA_ERROR();
    bs = 8;
    icg_rf_oobe_kernel<<<dim3(divUp(num_samples, bs), divUp(num_trees, bs),1), dim3(bs, bs, 1)>>>( 
        tree_adress_dmp.getBuffer(), single_tree_pitch,
        affiliation_dmp.getBuffer(), affiliation_dmp.stride[0],
        oobe_prediction_dmp.getBuffer(), oobe_prediction_dmp.stride[0], tp);
    ICG_RF_CHECK_CUDA_ERROR();
    Cuda::HostMemoryHeap<int,2> oobe_prediction_hmh(oobe_prediction_dmp);
    Cuda::HostMemoryHeap<int,1> labels_hmh(labels);
    int num_oob_samples = 0;
    int num_oob_valids = 0;
    for (int tree = 0; tree < num_trees; tree++) {
        for (int sample = 0; sample < num_samples; sample++) {
            int prediction = cudaGet(oobe_prediction_hmh, tree, sample);
            if (prediction != -1){
                num_oob_samples++;
                if (prediction == labels_hmh.getBuffer()[sample])
                    num_oob_valids++;
            }
        }
    }
    (*oobe) = 1.0f - (float) num_oob_valids / num_oob_samples;*/

 
    // copy trees to forest and free ressources
    for (int tree = 0; tree < num_trees; tree++) {
        float *forest_buffer = &(*forest)->getBuffer()[tree * num_tree_cols];
        hipMemcpy2D(forest_buffer, (*forest)->stride[0] * sizeof(float), 
            tree_adress_hmh.getBuffer()[tree], single_tree_pitch * sizeof(float),
            num_tree_cols * sizeof(float), cudaGet(nodes_per_level_hmh, tree, max_depth),
            hipMemcpyDeviceToDevice);
        hipFree(tree_adress_hmh.getBuffer()[tree]);
        ICG_RF_CHECK_CUDA_ERROR();
    }  

    hipUnbindTexture(input_tex);
    hipUnbindTexture(labels_tex);
    return true;
}


//-----------------------------------------------------------------------------
//! Evaluate the result of a random forest for a given data input. 
//! @param[in] forest The random forest to evaluate for. Its structure:
//!
//!      tree_idx | node_idx | is_terminal | m_i | w_i | theta | conf_j | label
//!
//!       tree_idx        Index of the tree in the forest, starting with '0'
//!       node_idx        Index of the node within a tree, starting with '0' 
//!       is_terminal     Specifies (0/1) whether a node or its parent is terminal
//!       m_i,w_i, theta  Specify the node hypotheses: m_1*w_1 + m_2* w_2 ... > theta
//!       conf_j          The confidence for the jth class in this node
//!       label           The label of the class with the highest confidence
//!
//! @param[in] input The input data to be classified (rows: datapoints, cols: features)
//! @param[out] confidences Confidences for each class, sized [num_classes x num_samples]
//! @param[out] predictions Class index with highest confidence, sized [1 x num_samples]
//! @param[in] num_classes The number of different classes (j)
//! @param[in] num_trees The number of trees in the forest 
//! @param[in] max_depth The depth of each tree in the forest
//! @param[in] num_features The number of features from which a hypothesis is generated (i)
//! @param[in] use_soft_voting If true, the confidences of all trees are used to predict labels.
//!            Otherwise, only the predictions of all trees are taken into account 
//! @return True on successful termination
bool cudaEvaluateForest(const Cuda::Array<float,2>& forest, 
                        const Cuda::Array<float,2>& input,
                        Cuda::DeviceMemoryPitched<float,2>* confidences,
                        Cuda::DeviceMemoryLinear<float,1>* predictions,
                        const int num_samples, const int num_trees, const int max_depth, const int num_tree_cols,
                        const int num_classes, const int num_features, const bool use_soft_voting)
{    
    if (num_samples != input.size[1])
        ICG_RF_THROW_ERROR("The input dimensions do not fit"); 

    if (confidences->size[0] != num_samples || confidences->size[1] != num_classes) 
        ICG_RF_THROW_ERROR("The confidences output dimensions do not fit"); 

    if (predictions->size[0] != num_samples) 
        ICG_RF_THROW_ERROR("The predictions output dimensions do not fit");

    TrainParams tp;
    tp.num_trees = num_trees;
    tp.tree_level = 0;
    tp.max_depth = max_depth;
    tp.num_samples = num_samples;
    tp.num_features = num_features;
    tp.num_classes = num_classes;
    tp.num_tree_cols = num_tree_cols;

    // Init forest texture
    forest_tex.filterMode = hipFilterModePoint;
    forest_tex.normalized = false;
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    hipBindTextureToArray(forest_tex, forest.getArray(), channelDesc);

    // Init samples texture
    samples_tex.filterMode = hipFilterModePoint;
    samples_tex.normalized = false;
    hipBindTextureToArray(samples_tex, input.getArray(), channelDesc);

    if (use_soft_voting) {
        int eval_block_size = 16;
        dim3 dim_block_eval(eval_block_size, eval_block_size, 1);
        dim3 dim_grid_eval(divUp(num_samples, eval_block_size), divUp(num_trees, eval_block_size),1);

        // compute confidences
        Cuda::DeviceMemoryPitched<float,2> buffer(Cuda::Size<2>(num_trees * num_classes, num_samples));
        icg_rf_eval_soft_kernel<<<dim_grid_eval, dim_block_eval>>>(            
            buffer.getBuffer(), buffer.stride[0], tp);
        ICG_RF_CHECK_CUDA_ERROR();

        buffer_tex.filterMode = hipFilterModePoint;
        buffer_tex.normalized = false;
        hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();

        // mean over confidences and compute predictions
        int pred_block_size = 128;
        dim3 dim_block_pred(pred_block_size, 1, 1);
        dim3 dim_grid_pred(divUp(num_samples, pred_block_size), 1, 1);

        Cuda::Array<float,2> buffer_arr(buffer); 
        hipBindTextureToArray(buffer_tex, buffer_arr.getArray(), channelDesc);       

        icg_rf_soft_pred_kernel<<<dim_grid_pred, dim_block_pred>>>(
            confidences->getBuffer(), confidences->stride[0],
            predictions->getBuffer(), tp);
        ICG_RF_CHECK_CUDA_ERROR();        

        hipUnbindTexture(buffer_tex);              
        hipDeviceSynchronize();      
    }
    else { // use hard voting
        // DEPRECATED
        ICG_RF_THROW_ERROR("Hard evaluation is deprecated. Call Jakob\n");  
        /*
        if (num_classes > 10) 
        ICG_RF_THROW_ERROR("The maximum number of classes is exceeded");    
        // get votings 
        int eval_block_size = 16; 
        dim3 dim_block_eval(eval_block_size, eval_block_size, 1);
        dim3 dim_grid_eval(divUp(num_samples, eval_block_size), divUp(num_trees, eval_block_size), 1);

        Cuda::DeviceMemoryPitched<float,2> buffer(Cuda::Size<2>(num_samples, num_trees));

        icg_rf_eval_hard_kernel<<<dim_grid_eval, dim_block_eval>>>(            
        buffer.getBuffer(), buffer.stride[0], tp);
        ICG_RF_CHECK_CUDA_ERROR();

        // mean over votings and compute predictions
        int pred_block_size = 64; 
        dim3 dim_block_pred(pred_block_size, 1, 1);
        dim3 dim_grid_pred(divUp(num_samples, pred_block_size), 1, 1);

        Cuda::Array<float,2> buffer_arr(buffer); 
        hipBindTextureToArray(buffer_tex, buffer_arr.getArray(), channelDesc);      

        icg_rf_hard_pred_kernel<<<dim_grid_pred, dim_block_pred>>>(
        confidences->getBuffer(), confidences->stride[0],
        predictions->getBuffer(), tp);
        ICG_RF_CHECK_CUDA_ERROR();

        hipUnbindTexture(buffer_tex);              
        hipDeviceSynchronize();  
        */
    }
    hipUnbindTexture(forest_tex);
    hipUnbindTexture(samples_tex);
    return true;
}

#endif //ICG_RF_FOREST_CU_
