#include "icg_rf_interface.cuh"
#include "icg_rf_forest.cu"

//-----------------------------------------------------------------------------
bool iTrainForest(const Cuda::Array<float,2>& input,
                  const Cuda::DeviceMemoryLinear<int,1>& labels,
                  const Cuda::DeviceMemoryLinear<float,1>& weights,
                  Cuda::DeviceMemoryPitched<float,2>** forest,
                  float* oobe,
                  const int num_samples, const int num_classes, 
                  const int num_trees, const int max_depth, const int num_tree_cols, 
                  const int num_hypotheses, const int num_features,
                  const float bag_ratio)
{
  return cudaTrainForest(input, labels, weights, forest, oobe,
    num_samples, num_classes, num_trees, max_depth, num_tree_cols,  
    num_hypotheses, num_features, bag_ratio);
}

//-----------------------------------------------------------------------------
bool iEvaluateForest(const Cuda::Array<float,2>& forest, 
                     const Cuda::Array<float,2>& input,
                     Cuda::DeviceMemoryPitched<float,2>* confidences,
                     Cuda::DeviceMemoryLinear<float,1>* predictions,
                     const int num_samples, const int num_trees, const int max_depth, const int num_tree_cols,
                     const int num_classes, const int num_features, const bool use_soft_voting) 
{
  return cudaEvaluateForest(
    forest, input, confidences, predictions, 
    num_samples, num_trees, max_depth, num_tree_cols,  
    num_classes, num_features, use_soft_voting);
}

