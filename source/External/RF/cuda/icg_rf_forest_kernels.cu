#include "hip/hip_runtime.h"
#ifndef ICG_RF_FOREST_KERNELS_CU_
#define ICG_RF_FOREST_KERNELS_CU_

#define LSON_IDX 0
#define OVERHEAD_OFS 1
#define FEAT_IDX(x) OVERHEAD_OFS + x * 2 
#define WEIGHT_IDX(x) OVERHEAD_OFS + x * 2 + 1
#define THRESH_IDX OVERHEAD_OFS + tp.num_features * 2 
#define CONF_IDX(x) OVERHEAD_OFS + x
#define LABEL_IDX OVERHEAD_OFS + tp.num_classes


////////////////////////////////////////////////////////////////////////////////
//! Training Kernels
////////////////////////////////////////////////////////////////////////////////
texture<float, 2, hipReadModeElementType> input_tex;
texture<int, 1, hipReadModeElementType> labels_tex;
texture<float, 2, hipReadModeElementType> r_weights_tex;
texture<int, 2, hipReadModeElementType> r_features_tex;

struct TrainParams {
    int num_trees; 
    unsigned char tree_level; 
    unsigned char max_depth; 
    int num_samples; 
    unsigned char num_features; 
    unsigned char num_classes;
    unsigned char num_hyp;
    int num_hyp_2;
    int num_tree_cols;
};

//-----------------------------------------------------------------------------
// \param buffer    This structure is initialized with 
template <typename T>
__global__ void icg_rf_init_kernel(T* buffer, const size_t cols, 
                                   const size_t rows, const size_t buffer_p,
                                   const T value)
{
    unsigned int col = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int row = blockIdx.y*blockDim.y + threadIdx.y;
    if (col < cols && row < rows)
        buffer[row * buffer_p + col] = value;
}

//-----------------------------------------------------------------------------
// \param bags      Bagging output, [num_trees x num_bag_samples] (w x h), holds for 
//                  every tree the indices of the samples within its bag
// \param affiliation   [num_trees x num_samples] (w x h), stores the node 
//                          index in every tree that each sample is currently
//                          falling in.
__global__ void icg_rf_bagging_kernel(int* bags, const size_t bags_p,
                                      int* affiliation, const size_t affiliation_p,
                                      const int num_trees, const int num_bag_samples)
{
    unsigned int tree_index = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int sample_index = blockIdx.y*blockDim.y + threadIdx.y;
    if (sample_index < num_bag_samples && tree_index < num_trees) {
        int index = (int)bags[sample_index * bags_p + tree_index];
        affiliation[index * affiliation_p + tree_index] = 0;
    }
}

//-----------------------------------------------------------------------------
// Checks for every leaf node of every tree, whether it is 
//      1: a node, which becomes a leaf at this tree level
//      2: a node, which needs to be split at this level
//      3: a node, which has leaf node parents at any level
// \param forest    the forest structure
// \param affiliation   [num_trees x num_samples] (w x h), stores the node 
//                          index in every tree that each sample is currently
//                          falling in.
// \param node_type     [num_trees x num_leaf_nodes] (w x h), stores the type
//                      (1,2 or 3) of all leafs of all trees
__global__ void icg_rf_classify_leaf_nodes_kernel(
    const int* num_nodes_per_level, const size_t num_nodes_per_level_p,
    int* affiliation, const size_t affiliation_p,
    int* node_type, const size_t node_type_p,
    const TrainParams tp)
{
    unsigned int tree_idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (tree_idx < tp.num_trees){
        unsigned int thread_leaf_index = blockIdx.y*blockDim.y + threadIdx.y;
        unsigned int leaf_idx = thread_leaf_index; 
        if (tp.tree_level > 0)
            leaf_idx = num_nodes_per_level[(tp.tree_level - 1) * num_nodes_per_level_p + tree_idx] + thread_leaf_index; 
        unsigned int leaf_idx_stop = num_nodes_per_level[tp.tree_level * num_nodes_per_level_p + tree_idx];

        if (leaf_idx < leaf_idx_stop) {
            // check whether this node is a split node
            bool is_split_node = false;
            if (tp.tree_level != tp.max_depth) {
                bool first_node_found = false;
                int first_label =  -1;                
                for (int sample = 0; sample < tp.num_samples; sample++){
                    if (affiliation[affiliation_p * sample + tree_idx] == leaf_idx) {
                        if (!first_node_found) 
                            first_label = tex1Dfetch(labels_tex,sample);
                        else 
                            if (first_label != tex1Dfetch(labels_tex,sample))
                                is_split_node = true; // this node contains samples of different labels -> split node 
                        first_node_found = true;                    
                    }
                }
            }
            int type = 1;
            if (is_split_node)
                type = 2;

            node_type[thread_leaf_index * node_type_p + tree_idx] = type;
        }
    }
}

//-----------------------------------------------------------------------------
// Assigns the feature indices to the forest structure
// \param forest    the forest structure
// \param split_subset  [num_split_nodes x 2] (w x h), stores in the first row
//                      the tree index, and in the second row the node index of
//                      a split node
__global__ void icg_rf_split_assign_feature_kernel(
    float** trees, const size_t tree_p,
    const int* split_subset, const size_t num_splits, const size_t split_subset_p, 
    const TrainParams tp)
{
    unsigned int split_idx = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int feat_idx = blockIdx.y*blockDim.y + threadIdx.y;

    if (split_idx < num_splits && feat_idx < tp.num_features) {
        unsigned int tree_idx = split_subset[split_idx];
        float* tree = trees[tree_idx];
        unsigned int node_idx = split_subset[split_subset_p + split_idx];
        
        // assign the feature indices directly to the forest
        int feature = tex2D(r_features_tex, split_idx, feat_idx);
        // assign feature index to forest
        tree[tree_p * node_idx + FEAT_IDX(feat_idx)] = feature;
        // assign lson index
        tree[tree_p * node_idx + LSON_IDX] = split_subset[split_subset_p * 2 + split_idx]; //FIXXME not good here
    }
}

//-----------------------------------------------------------------------------
// Computes the minimum and maximum threshold for every split node at the 
// current level.
// \param split_subset  [num_split_nodes x 2] (w x h), stores in the first row
//                      the tree index, and in the second row the node index of
//                      a split node
// \param affiliation   [num_trees x num_samples] (w x h), stores the node 
//                          index in every tree that each sample is currently
//                          falling in.
// \param thresh_limits     [num_split_nodes x num_hypotheses * 2] (w x h),
//                          stores the minimum and maximum response of all 
//                          samples in the current split node for all weight
//                          hypotheses.
__global__ void icg_rf_split_compute_thresh_kernel(
    const int* split_subset, const size_t num_splits, const size_t split_subset_p, 
    const int* affiliation, const size_t affiliation_p,
    float* thresh_limits, const size_t thresh_limits_p,
    const TrainParams tp)
{
    unsigned int split_idx = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int hyp_idx = blockIdx.y*blockDim.y + threadIdx.y;

    if (split_idx < num_splits && hyp_idx < tp.num_hyp) {
        unsigned int tree_idx = split_subset[split_idx];
        unsigned int node_idx = split_subset[split_subset_p + split_idx];

        // init the limits for this node 
        float thresh_min = 1e10;
        float thresh_max = -1e10;        

        // compute the limits from the responses of every affiliated sample
        for (int sample = 0; sample < tp.num_samples; sample++){ 
            if (affiliation[affiliation_p * sample + tree_idx] == node_idx) {
                // calulate threshold 
                float response = 0;
                for (unsigned char feature = 0; feature < tp.num_features; feature++){
                    int feature_index = tex2D(r_features_tex, split_idx, feature);
                    float xi = tex2D(input_tex, feature_index, sample);                               
                    float wi = tex2D(r_weights_tex, split_idx, hyp_idx * tp.num_features + feature);
                    response += xi*wi;                        
                } 
                if (response < thresh_min)
                    thresh_min = response;
                if (response > thresh_max)
                    thresh_max = response;
            }                        
        } 
        // assign threshold limits
        thresh_limits[thresh_limits_p * (hyp_idx * 2) + split_idx] = thresh_min;
        thresh_limits[thresh_limits_p * (hyp_idx * 2 + 1) + split_idx] = thresh_max;        
    }
}

//-----------------------------------------------------------------------------
// \param split_subset  [num_split_nodes x 2] (w x h), stores in the first row
//                      the tree index, and in the second row the node index of
//                      a split node
// \param weights       [num_samples], stores a sample weight
// \param affiliation   [num_trees x num_samples] (w x h), stores the node 
//                          index in every tree that each sample is currently
//                          falling in.
// \param thresh_limits     [num_split_nodes x num_hypotheses * 2] (w x h),
//                          stores the minimum and maximum response of all 
//                          samples in the current split node for all weight
//                          hypotheses.
// \param pk_l, pk_r    [num_split_nodes x num_hypotheses^2 * num_classes] (w x h),
//                      store the number of samples falling in the left and 
//                      right child node for every hypothesis and every threshold
__global__ void icg_rf_split_eval_thresh_kernel(
    const int* split_subset, const size_t num_splits, const size_t split_subset_p, 
    const float* weights,
    const int* affiliation, const size_t affiliation_p,
    float* thresh_limits, const size_t thresh_limits_p,
    float* pk_l, const size_t pk_l_p, float* pk_r, const size_t pk_r_p,
    const TrainParams tp)
{
    unsigned int split_idx = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int hyp_idx = blockIdx.y*blockDim.y + threadIdx.y;

    if (split_idx < num_splits && hyp_idx < tp.num_hyp) {
        unsigned int tree_idx = split_subset[split_idx];
        unsigned int node_idx = split_subset[split_subset_p + split_idx];
        
        float min_thresh = thresh_limits[thresh_limits_p * (hyp_idx * 2) + split_idx];
        float max_thresh = thresh_limits[thresh_limits_p * (hyp_idx * 2 + 1) + split_idx];
        float thresh_step = (max_thresh - min_thresh)/tp.num_hyp;
                            
        // evaluate node for each threshold to find best split                       
        for (int sample = 0; sample < tp.num_samples; sample++){  
            if (affiliation[affiliation_p * sample + tree_idx] == node_idx) {
                // compute this hypothesis
                float sum = 0;
                for (unsigned char feature = 0; feature < tp.num_features; feature++){
                    int feature_index = tex2D(r_features_tex, split_idx, feature);
                    float xi = tex2D(input_tex, feature_index, sample);
                    float wi = tex2D(r_weights_tex, split_idx, hyp_idx * tp.num_features + feature);
                    sum += xi*wi;
                }
                // evaluate all thresholds
                unsigned char label = tex1Dfetch(labels_tex,sample);
                float weight = weights[sample]; 
                for (unsigned char thresh_hyp = 0; thresh_hyp < tp.num_hyp; thresh_hyp++) {
                    float thresh = min_thresh + (thresh_hyp + 0.5f) * thresh_step; 
                    if (sum > thresh) 
                        pk_l[pk_l_p * (tp.num_hyp_2 * label + hyp_idx * tp.num_hyp + thresh_hyp) + split_idx] += weight;
                    else 
                        pk_r[pk_r_p * (tp.num_hyp_2 * label + hyp_idx * tp.num_hyp + thresh_hyp) + split_idx] += weight;

                }                       
            }
        }
    }
}

//-----------------------------------------------------------------------------
// Examinates the structures pkl and pkr filled by icg_rf_split_eval_thresh_kernel
// and stores the best split by evaluating a Gini function over all hypotheses.
// \param forest    the forest structure
// \param split_subset  [num_split_nodes x 2] (w x h), stores in the first row
//                      the tree index, and in the second row the node index of
//                      a split node
// \param thresh_limits     [num_split_nodes x num_hypotheses * 2] (w x h),
//                          stores the minimum and maximum response of all 
//                          samples in the current split node for all weight
//                          hypotheses.
// \param pk_l, pk_r    [num_split_nodes x num_hypotheses^2 * num_classes] (w x h),
//                      store the number of samples falling in the left and 
//                      right child node for every hypothesis and every threshold
__global__ void icg_rf_split_select_best_kernel(
    float** trees, const size_t tree_p,
    const int* split_subset, const size_t num_splits, const size_t split_subset_p,
    float* thresh_limits, const size_t thresh_limits_p,
    float* pk_l, const size_t pk_l_p, float* pk_r, const size_t pk_r_p,
    const TrainParams tp)
{
    unsigned int split_idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (split_idx < num_splits) {
        unsigned int tree_idx = split_subset[split_idx];
        float* tree = trees[tree_idx];
        unsigned int node_idx = split_subset[split_subset_p + split_idx];        

        // compute scores for the child nodes
        int min_index = 0;
        float min_score = 0.5f;
        for (int i = 0; i < tp.num_hyp_2; i++) {
            float elements_l = 0;
            float elements_r = 0;
            for (unsigned char label = 0; label < tp.num_classes; label++) {
                elements_l += pk_l[pk_l_p * (tp.num_hyp_2 * label + i) + split_idx];
                elements_r += pk_r[pk_r_p * (tp.num_hyp_2 * label + i) + split_idx];
            }            
            float gini_l = 0;
            float gini_r = 0;
            for (unsigned char label = 0; label < tp.num_classes; label++) {
                float pkl = pk_l[pk_l_p * (tp.num_hyp_2 * label + i) + split_idx] / elements_l;
                float pkr = pk_r[pk_r_p * (tp.num_hyp_2 * label + i) + split_idx] / elements_r;
                gini_l += pkl * (1 - pkl); 
                gini_r += pkr * (1 - pkr);
            }
            float score = (elements_l * gini_l + elements_r * gini_r) / (elements_l + elements_r);
            if (score < min_score){
                min_score = score;
                min_index = i;
            }
        }

        // find best hypothesis and best threshold
        unsigned char best_hyp_index = min_index / tp.num_hyp;
        unsigned char best_thresh_index = min_index % tp.num_hyp;

        // assign best weights to forest
        for (unsigned char i = 0; i < tp.num_features; i++){
            float wi = tex2D(r_weights_tex, split_idx, best_hyp_index * tp.num_features + i);
            tree[tree_p * node_idx + WEIGHT_IDX(i)] = wi;
        }

        // assign threshold to forest
        float min_thresh = thresh_limits[thresh_limits_p * (best_hyp_index * 2) + split_idx];
        float max_thresh = thresh_limits[thresh_limits_p * (best_hyp_index * 2 + 1) + split_idx];
        float thresh_step = (max_thresh - min_thresh)/tp.num_hyp;
        tree[tree_p * node_idx + THRESH_IDX] = min_thresh + (best_thresh_index + 0.5f ) * thresh_step;
    }
}

//-----------------------------------------------------------------------------
// Computes the confidences for each leaf node using the amount of samples of
// each class falling into that node.
// \param forest    the forest structure
// \param weights       [num_samples], stores a sample weight
// \param split_subset  [num_split_nodes x 2] (w x h), stores in the first row
//                      the tree index, and in the second row the node index of
//                      a split node
// \param affiliation   [num_trees x num_samples] (w x h), stores the node 
//                          index in every tree that each sample is currently
//                          falling in.
// \param node_type     [num_trees x num_leaf_nodes] (w x h), stores the type
//                      (1,2 or 3) of all leafs of all trees

__global__ void icg_rf_train_leaf_node_kernel(
    float** trees, const size_t tree_p,
    const float* weights, 
    const int* leaf_subset, const size_t num_leafs, const size_t leaf_subset_p, 
    const int* affiliation, const size_t affiliation_p,
    const TrainParams tp)
{

    unsigned int split_idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (split_idx < num_leafs) {
        unsigned int tree_idx = leaf_subset[split_idx];
        float* tree = trees[tree_idx];
        unsigned int node_idx = leaf_subset[leaf_subset_p + split_idx];

        extern __shared__ int accu[];
        unsigned int accu_offset = threadIdx.x * tp.num_classes;
        for (int i = 0; i < tp.num_classes; i++)
          accu[accu_offset + i] = 0.0f;

        float total_weight = 0.0f;
        for (int sample = 0; sample < tp.num_samples; sample++){ 
            if (affiliation[affiliation_p * sample + tree_idx] == node_idx){
                unsigned char label = tex1Dfetch(labels_tex,sample);
                float weight = weights[sample];
                accu[accu_offset + label] += weight;
                total_weight += weight;
            }
        }
        // assign confidences and label
        if (total_weight > 0.0f) {
            int max_class = 0;
            int max_count = 0;
            for (unsigned char i = 0; i < tp.num_classes; i++) {
                tree[tree_p * node_idx + CONF_IDX(i)] = (float) accu[accu_offset + i] / total_weight;
                if (accu[accu_offset + i] > max_count) {
                    max_class = i;
                    max_count = accu[accu_offset + i]; 
                }
            }
            tree[tree_p * node_idx + LABEL_IDX] = max_class;
        }
        else {
            // due to numerical instabilities it can happen, that a sample falls directly on a threshold during 
            // score computation. Therefore it can (unlikely) happen that a node gets created, but no sample falls in
            // it during evaluation -> so we need to catch this case by inserting zero confidences...
            for (unsigned char i = 0; i < tp.num_classes; i++) 
                tree[tree_p * node_idx + CONF_IDX(i)] = 0.0f;
            tree[tree_p * node_idx + LABEL_IDX] = 0;
        }
        // assign left son's index
        tree[tree_p * node_idx + LSON_IDX] = -1;
    }
}

//-----------------------------------------------------------------------------
__global__ void icg_rf_eval_affinity_kernel(
    float** trees, const size_t tree_p,
    int *affiliation, size_t affiliation_p, TrainParams tp)
{
    unsigned int sample_index = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int tree_index = blockIdx.y*blockDim.y + threadIdx.y;  

    if (sample_index < tp.num_samples && tree_index < tp.num_trees) {
        if (affiliation[affiliation_p * sample_index + tree_index] != -1) {
            float* tree = trees[tree_index];
            // set initial index in tree
            unsigned int node_index = 0;
            // go through whole tree
            for (unsigned char level = 0; level < tp.max_depth; level++) {
                int left_son_index = tree[tree_p * node_index + LSON_IDX];
                // break if terminal node has been found 
                if (left_son_index < 1)
                    break;
                // evaluate at this node
                float sum = 0;
                for (unsigned char feature = 0; feature < tp.num_features; feature++) {                    
                    int feature_index = (int) tree[tree_p * node_index + FEAT_IDX(feature)];
                    float wi = tree[tree_p * node_index + WEIGHT_IDX(feature)];
                    float xi = tex2D(input_tex, feature_index, sample_index);
                    sum += xi * wi;
                }
                float theta = tree[tree_p * node_index + THRESH_IDX];
                // recompute the node_index
                if (sum < theta) 
                    node_index = left_son_index; // index of the left son                                       
                else 
                    node_index = left_son_index + 1; // index of the right son
            }
            // store only the current node index
            affiliation[affiliation_p * sample_index + tree_index] = node_index;
        }
    }
}

//-----------------------------------------------------------------------------
__global__ void icg_rf_oobe_kernel(
    float** trees, const size_t tree_p,
    int *affiliation, size_t affiliation_p, 
    int *prediction, size_t prediction_p, TrainParams tp)
{
    unsigned int sample_index = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int tree_index = blockIdx.y*blockDim.y + threadIdx.y;  

    if (sample_index < tp.num_samples && tree_index < tp.num_trees) {
        if (affiliation[affiliation_p * sample_index + tree_index] == -1) {
            float* tree = trees[tree_index];
            // set initial index in tree
            unsigned int node_index = 0;
            // go through whole tree
            for (unsigned char level = 0; level < tp.max_depth; level++) {
                int left_son_index = tree[tree_p * node_index + LSON_IDX];
                // break if terminal node has been found 
                if (left_son_index < 1)
                    break;
                // evaluate at this node
                float sum = 0;
                for (unsigned char feature = 0; feature < tp.num_features; feature++) {                    
                    int feature_index = (int) tree[tree_p * node_index + FEAT_IDX(feature)];
                    float wi = tree[tree_p * node_index + WEIGHT_IDX(feature)];
                    float xi = tex2D(input_tex, feature_index, sample_index);
                    sum += xi * wi;
                }
                float theta = tree[tree_p * node_index + THRESH_IDX];
                // recompute the node_index
                if (sum < theta) 
                    node_index = left_son_index; // index of the left son                                       
                else 
                    node_index = left_son_index + 1; // index of the right son
            }
            // store only the current node index
            prediction[prediction_p * sample_index + tree_index] = tree[tree_p * node_index + LABEL_IDX];
        }
    }
}

////////////////////////////////////////////////////////////////////////////////
//! Evaluation Kernels
////////////////////////////////////////////////////////////////////////////////
texture<float, 2, hipReadModeElementType> forest_tex;
texture<float, 2, hipReadModeElementType> samples_tex;
texture<float, 2, hipReadModeElementType> buffer_tex;

//-----------------------------------------------------------------------------
//DEPRECATED
/*__global__ void icg_rf_eval_hard_kernel(float* buffer, size_t buffer_p,
                                        TrainParams tp)
{
    unsigned int sample_index = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int tree_index = blockIdx.y*blockDim.y + threadIdx.y;

    if (sample_index < tp.num_samples && tree_index < tp.num_trees) {
        // calculate the numbers of elements in a tree
        unsigned int tree_offset = tree_index * tp.num_tree_cols;
        // set initial index in tree
        unsigned int node_index = 0;
        // go through whole tree
        for (int level = 0; level < tp.max_depth; level++) {
            // break if terminal node has been found
            if (tex2D(forest_tex, tree_offset, node_index))
                break;
            // evaluate at this node
            float sum = 0;
            for (int feature = 0; feature < tp.num_features; feature++) {
                int feature_index = (int) tex2D(forest_tex, tree_offset + 1 + feature * 2, node_index);
                float mi = tex2D(samples_tex, feature_index, sample_index);
                float wi = tex2D(forest_tex, tree_offset + 1 + feature * 2 + 1, node_index);
                sum += mi * wi;
            }
            float theta = tex2D(forest_tex, tree_offset + 1 + tp.num_features * 2, node_index);
            // recompute the node_index
            if (sum < theta) 
                node_index = node_index * 2 + 1; // index of the left son                                       
            else 
                node_index = node_index * 2 + 2; // index of the right son
        }

        // increase the voting for the predicted class
        int label = tex2D(forest_tex, tree_offset + tp.num_tree_cols - 1, node_index);  
        buffer[tree_index * buffer_p + sample_index] = label;    
    }
}
//-----------------------------------------------------------------------------
__global__ void icg_rf_hard_pred_kernel(float *confidences, size_t confidences_p,
                                        float *predictions, 
                                        TrainParams tp)
{
    unsigned int sample_index = blockIdx.x*blockDim.x + threadIdx.x; 

    if (sample_index < tp.num_samples) {
        int classes[10] = {0,0,0,0,0,0,0,0,0,0};
        // accumulate labels
        for (int tree = 0; tree < tp.num_trees; tree++){
            unsigned char label = tex2D(buffer_tex, sample_index, tree);
            classes[label]++;
        }
        // assign predictions and confidences
        float max_conf = 0;
        int max_class = 0;
        for (int label = 0; label < tp.num_classes; label++){
            float act_conf = (float)classes[label] / tp.num_trees;
            confidences[label * confidences_p + sample_index] = act_conf;
            if (act_conf > max_conf) {
                max_conf = act_conf;
                max_class = label;
            }
        }
        predictions[sample_index] = max_class;
    }  
}*/

//-----------------------------------------------------------------------------
__global__ void icg_rf_eval_soft_kernel(float *output, size_t output_p,
                                        TrainParams tp)
{
    unsigned int sample_index = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int tree_index = blockIdx.y*blockDim.y + threadIdx.y;  

    if (sample_index < tp.num_samples && tree_index < tp.num_trees) {  
        // calculate the numbers of elements in a tree
        unsigned int tree_offset = tree_index * tp.num_tree_cols;
        // set initial index in tree
        unsigned int node_index = 0;
        // go through whole tree
        for (int level = 0; level < tp.max_depth; level++) {
            int left_son_index = tex2D(forest_tex, tree_offset + LSON_IDX, node_index);
            // break if terminal node has been found
            if (left_son_index < 0)
                break;
            // evaluate at this node
            float sum = 0;
            for (int feature = 0; feature < tp.num_features; feature++) {
                int feature_index = (int) tex2D(forest_tex, tree_offset + FEAT_IDX(feature), node_index);
                float wi = tex2D(forest_tex, tree_offset + WEIGHT_IDX(feature), node_index);
                float mi = tex2D(samples_tex, feature_index, sample_index);
                sum += mi * wi;
            }
            float theta = tex2D(forest_tex, tree_offset + THRESH_IDX, node_index);
            // recompute the node_index
            if (sum < theta) 
                node_index = left_son_index; // index of the left son                                       
            else 
                node_index = left_son_index + 1; // index of the right son
        }

        // return the confidences for each class 
        for (int label = 0; label < tp.num_classes; label++) {
            float confidence = tex2D(forest_tex, tree_offset + CONF_IDX(label), node_index); 
            output[sample_index * output_p + tree_index * tp.num_classes + label] = confidence;        
        }
    }
}

//-----------------------------------------------------------------------------
__global__ void icg_rf_soft_pred_kernel(float *confidences, size_t confidences_p,
                                        float *predictions, 
                                        TrainParams tp)
{
    unsigned int sample_index = blockIdx.x*blockDim.x + threadIdx.x; 

    if (sample_index < tp.num_samples) {
        float max_confidence = 0;
        float max_label = 0;
        for (int label = 0; label < tp.num_classes; label++) {
            // sum over the confidences of all trees
            float sum = 0;
            for (int tree = 0; tree < tp.num_trees; tree++){
                sum += tex2D(buffer_tex, tree * tp.num_classes + label, sample_index);      
            }
            float confidence = sum / tp.num_trees;
            confidences[label * confidences_p + sample_index] = confidence;
            if (confidence > max_confidence){
                max_confidence = confidence;
                max_label = label;
            }
        }
        predictions[sample_index] = max_label;
    }  
}

#endif //ICG_RF_FOREST_KERNELS_CU_
